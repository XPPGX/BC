#include "hip/hip_runtime.h"
#ifndef COMMON
#define COMMON
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#endif

#include "headers.h"

#define CHECK(call){                                                           \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr,"Error: %s:%d,",__FILE__,__LINE__);                 \
        fprintf(stderr,"code: %d,reason: %s\n",error,                      \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

struct Single_values{
    //already known
    int newEndID;
    int oriEndNodeID;
    int csrVSize;
    int csrESize;

    //get the values when traversing
    int Q_front; // a variable point to the front end of the g_Q
    int Q_next_front; //just a variable point to the front end of the g_Q in next iteration
    int Q_rear; //just a variable point to the rear end of the g_Q
    int mappingCount;
};

void quicksort_nodeID_with_degree(int* _nodes, int* _nodeDegrees, int _left, int _right);
void sortEachComp_NewID_with_degree(struct CSR* _csr, int* _newNodesID_arr, int* _newNodesDegree_arr);
__global__ void g_newID_infos_AOS_to_SOA(struct newID_info* _g_newID_infos_AOS, int* _g_newID_infos_ff, int* _g_newID_infos_r, struct Single_values* _g_csr_values);
__global__ void resetData(int* _g_dist_s, int* _g_Q, unsigned int* _g_SI, unsigned int* _g_R, struct Single_values* _g_single_values, int sourceNewID);
__global__ void mappingCount(int* _g_orderedCsrV, int* _g_orderedCsrE, int* _g_mapNodeID_New_to_Old, int* _g_nodeDone, struct Single_values* _g_single_values, int sourceNewID);
__global__ void ordinaryTraverseOneLevel(int* _g_orderedCsrV, int* _g_orderedCsrE, int* _g_mapNodeID_New_to_Old, int* _g_dist_s, int* _g_Q, struct Single_values* _g_single_values, int _Q_size);
__global__ void contributeDistToEachNodes(int* _g_dist_s, int* _g_CCs, int* _g_mapNodeID_New_to_Old, int* _g_newID_infos_ff, int* _g_newID_infos_r, int* _g_comp_newCsrOffset, int* _g_newNodesCompID, int* _g_newNodesID_arr, int _comp_size, int _sourceNewID, struct Single_values* _g_Single_values);
__global__ void copySI_toDevice(int* _g_mapNodes, unsigned int* _g_SI, int _mappingCount);
/**
 * @brief
 * preprocess : sequential => D1, AP, rebuild.
 * traversing : parallel => source traversal, neighbor sharing.
*/
void preprocess_then_parallel_sharedBased_DegreeOrder(struct CSR* _csr, int* CCs){

    #pragma region Preprocess
    D1Folding(_csr);
    AP_detection(_csr);
    AP_Copy_And_Split(_csr);
    struct newID_info* newID_infos = rebuildGraph(_csr); //rebuild graph for better memory access speed
    const int oriEndNodeID = _csr->endNodeID - _csr->apCloneCount; //原本graph的endNodeID
    
    //Sort aliveNodeID with degree
    int* newNodesID_arr     = (int*)malloc(sizeof(int) * (_csr->newEndID + 1));
    int* newNodesDegree_arr = (int*)malloc(sizeof(int) * (_csr->newEndID + 1));
    sortEachComp_NewID_with_degree(_csr, newNodesID_arr, newNodesDegree_arr);

    //整理一些要傳進GPU的常數值
    struct Single_values* Single_values = (struct Single_values*)malloc(sizeof(struct Single_values));
    Single_values->csrVSize     = _csr->csrVSize;
    Single_values->csrESize     = _csr->csrESize;
    Single_values->newEndID     = _csr->newEndID;
    Single_values->oriEndNodeID = oriEndNodeID;

    int* mapNodes       = (int*)malloc(sizeof(int) * 32);
    unsigned int* SI    = (unsigned int*)calloc(sizeof(unsigned int), (_csr->csrVSize) * 2);
    unsigned int* R     = (unsigned int*)calloc(sizeof(unsigned int), (_csr->csrVSize) * 2);

    #pragma endregion Preprocess

    #pragma region copyDataToGPU
    //declare
    int* g_CCs;

    int* g_mapNodeID_New_to_Old;
    int* g_mapNodeID_Old_to_New;
    int* g_orderedCsrV;
    int* g_orderedCsrE;
    int* g_comp_newCsrOffset; 
    int* g_newNodesCompID;
    int* g_newNodesID_arr;
    struct Single_values* g_Single_values;
    
    int* g_newID_infos_ff;
    int* g_newID_infos_r;
    struct newID_info* g_newID_infos_AOS;
    
    int* g_nodeDone;
    int* g_dist_s;
    int* g_dist_w;
    int* g_Q;
    int* g_mapNodes;
    unsigned int* g_SI; //SI, sharedBitIndex
    unsigned int* g_R; //R, relation
    
    //malloc
    hipMalloc((void**)&g_CCs, sizeof(int) * (_csr->csrVSize));
    hipMalloc((void**)&g_mapNodeID_New_to_Old, sizeof(int) * (_csr->csrVSize) * 2);
    hipMalloc((void**)&g_mapNodeID_Old_to_New, sizeof(int) * (_csr->csrVSize) * 2);
    hipMalloc((void**)&g_orderedCsrV, sizeof(int) * (_csr->csrVSize) * 2);
    hipMalloc((void**)&g_orderedCsrE, sizeof(int) * (_csr->csrESize) * 4);
    hipMalloc((void**)&g_comp_newCsrOffset, sizeof(int) * _csr->aliveNodeCount);
    hipMalloc((void**)&g_newNodesCompID, sizeof(int) * _csr->csrVSize * 2);
    hipMalloc((void**)&g_newNodesID_arr, sizeof(int) * (_csr->newEndID + 1));
    hipMalloc((void**)&g_Single_values, sizeof(struct Single_values));

    hipMalloc((void**)&g_newID_infos_AOS, sizeof(struct newID_info) * (_csr->newEndID + 1));
    hipMalloc((void**)&g_newID_infos_ff, sizeof(int) * (_csr->newEndID + 1));
    hipMalloc((void**)&g_newID_infos_r, sizeof(int) * (_csr->newEndID + 1));

    hipMalloc((void**)&g_nodeDone, sizeof(int) * (_csr->csrVSize) * 2);
    hipMalloc((void**)&g_dist_s, sizeof(int) * (_csr->csrVSize) * 2);
    hipMalloc((void**)&g_dist_w, sizeof(int) * (_csr->csrVSize) * 2);
    hipMalloc((void**)&g_Q, sizeof(int) * (_csr->csrVSize) * 2);
    hipMalloc((void**)&g_mapNodes, sizeof(int) * 32);
    hipMalloc((void**)&g_SI, sizeof(unsigned int) * (_csr->csrVSize) * 2);
    hipMalloc((void**)&g_R, sizeof(unsigned int) * (_csr->csrVSize) * 2);
    //memory copy from host to GPU
    hipMemset(g_CCs, 0, sizeof(int) * (_csr->csrVSize)); //因為有些node是AP且已在sequential取得CC了，他們的g_CCs，在kernel計算完之後會是0。
    hipMemcpy(g_mapNodeID_New_to_Old, _csr->mapNodeID_New_to_Old, sizeof(int) * (_csr->csrVSize) * 2, hipMemcpyHostToDevice);
    hipMemcpy(g_mapNodeID_Old_to_New, _csr->mapNodeID_Old_to_new, sizeof(int) * (_csr->csrVSize) * 2, hipMemcpyHostToDevice);
    hipMemcpy(g_orderedCsrV, _csr->orderedCsrV, sizeof(int) * (_csr->csrVSize) * 2, hipMemcpyHostToDevice);
    hipMemcpy(g_orderedCsrE, _csr->orderedCsrE, sizeof(int) * (_csr->csrESize) * 4, hipMemcpyHostToDevice);
    hipMemcpy(g_comp_newCsrOffset, _csr->comp_newCsrOffset, sizeof(int) * (_csr->aliveNodeCount), hipMemcpyHostToDevice);
    hipMemcpy(g_newNodesCompID, _csr->newNodesCompID, sizeof(int) * (_csr->csrVSize) * 2, hipMemcpyHostToDevice);
    hipMemcpy(g_newNodesID_arr, newNodesID_arr, sizeof(int) * (_csr->newEndID + 1), hipMemcpyHostToDevice);
    hipMemcpy(g_Single_values, Single_values, sizeof(struct Single_values), hipMemcpyHostToDevice);

    //用一個 kenrel 去對 g_newID_infos_ff, g_newID_infos_r 賦值。
    hipMemcpy(g_newID_infos_AOS, newID_infos, sizeof(struct newID_info) * (_csr->newEndID + 1), hipMemcpyHostToDevice);
    g_newID_infos_AOS_to_SOA<<<(_csr->newEndID + 32 - 1 / 32), 32>>>(g_newID_infos_AOS, g_newID_infos_ff, g_newID_infos_r, g_Single_values);
    
    hipMemset(g_nodeDone, 0, sizeof(int) * (_csr->csrVSize) * 2);
    hipMemset(g_dist_s, -1, sizeof(int) * (_csr->csrVSize) * 2);
    hipMemset(g_dist_w, -1, sizeof(int) * (_csr->csrVSize) * 2);
    hipMemset(g_Q, 0, sizeof(int) * (_csr->csrVSize) * 2);
    hipMemset(g_mapNodes, 0, sizeof(int) * 32);
    hipMemset(g_SI, 0, sizeof(unsigned int) * (_csr->csrVSize) * 2);
    hipMemset(g_R, 0, sizeof(unsigned int) * (_csr->csrVSize) * 2);
    
    // int* check_ff_arr = (int*)malloc(sizeof(int) * (_csr->newEndID + 1));
    // int* check_r_arr = (int*)malloc(sizeof(int) * (_csr->newEndID + 1));
    // hipMemcpy(check_ff_arr, g_newID_infos_ff, sizeof(int) * (_csr->newEndID + 1), hipMemcpyDeviceToHost);
    // hipMemcpy(check_r_arr, g_newID_infos_r, sizeof(int) * (_csr->newEndID + 1), hipMemcpyDeviceToHost);
    // for(int i = 0 ; i <= _csr->newEndID ; i ++){
    //     if(check_ff_arr[i] != newID_infos[i].ff){printf("node %d.ff wrong\n", i);}
    //     if(check_r_arr[i] != newID_infos[i].w){printf("node %d.r wrong\n", i);}
    // }
    #pragma endregion copyDataToGPU

    #pragma region Traverse
    int* nodeDone = (int*)calloc(sizeof(int*), (_csr->csrVSize) * 2);
    for(int compID = 0 ; compID <= _csr->compEndID ; compID ++){
        // printf("comp %d nodes :\n", _csr->comp_newCsrOffset[compID + 1] - _csr->comp_newCsrOffset[compID]);
        // for(int newID_idx = _csr->comp_newCsrOffset[compID + 1] - 1 ; newID_idx >= _csr->comp_newCsrOffset[compID] ; newID_idx --){
        //     int newID = newNodesID_arr[newID_idx];
        //     printf("n[%d] = {", newID);
        //     for(int nID_idx = _csr->orderedCsrV[newID] ; nID_idx < _csr->orderedCsrV[newID + 1] ; nID_idx ++){
        //         int nID = _csr->orderedCsrE[nID_idx];
        //         printf("%d, ", nID);
        //     }
        //     printf("}\n");
        // }
        // printf("\n");


        for(int newID_idx = _csr->comp_newCsrOffset[compID + 1] - 1 ; newID_idx >= _csr->comp_newCsrOffset[compID] ; newID_idx --){
            int sourceNewID = newNodesID_arr[newID_idx];
            int sourceOldID = _csr->mapNodeID_New_to_Old[sourceNewID];
            
            /**
             * 不做：
             * 1. 已經 nodeDone = 1 的 node
             * 2. CloneAP (藉由 (sourceOldID > oriEndNodeID)判斷一個node是不是 CloneAP) 
            */
            if(nodeDone[sourceNewID] == 1 || (sourceOldID > oriEndNodeID)){
                continue;
            }
            int compSize = _csr->comp_newCsrOffset[compID + 1] - _csr->comp_newCsrOffset[compID];

            nodeDone[sourceNewID] = 1;

            Single_values->Q_front = 0;
            Single_values->Q_next_front = 1;
            Single_values->Q_rear = 1;
            
            resetData<<<((_csr->csrVSize * 2 + 1023)) / 1024, 1024>>>(g_dist_s, g_Q, g_SI, g_R, g_Single_values, sourceNewID);
            int sourceNewID_degree = _csr->orderedCsrV[sourceNewID + 1] - _csr->orderedCsrV[sourceNewID];
            mappingCount<<<(sourceNewID_degree + 95) / 96, 96>>>(g_orderedCsrV, g_orderedCsrE, g_mapNodeID_New_to_Old, g_nodeDone, g_Single_values, sourceNewID);
            int mappingCount = 0;
            hipMemcpy(&mappingCount, &(g_Single_values->mappingCount), sizeof(int), hipMemcpyDeviceToHost);
            printf("source.degree = %d, mappingCount = %d\n", sourceNewID_degree, mappingCount);
            hipDeviceSynchronize();

            mappingCount = 0 ; //[test] if(mappingCount < 3)
            int Q_size = 1; //means the queue size is 1 at the beginning
            int threadNum = 96; //可以用thread decide 1

            if(mappingCount < 3){ //
                

                while(Q_size > 0){
                    int blockNum = Q_size;
                    printf("[HI][1]\n");
                    hipMemcpy(g_Single_values, Single_values, sizeof(struct Single_values), hipMemcpyHostToDevice);
                    printf("[HI][2]\n");
                    ordinaryTraverseOneLevel<<<blockNum, threadNum>>>(g_orderedCsrV, g_orderedCsrE, g_mapNodeID_New_to_Old, g_dist_s, g_Q, g_Single_values, Q_size);
                    printf("[HI][3]\n");
                    hipMemcpy(Single_values, g_Single_values, sizeof(struct Single_values), hipMemcpyDeviceToHost);
                    printf("[HI][4]\n");
                    Single_values->Q_front = Single_values->Q_next_front;
                    Single_values->Q_next_front = Single_values->Q_rear;

                    Q_size = Single_values->Q_rear - Single_values->Q_front;
                    printf("h_Q_front = %d, h_Q_next_front = %d, h_Q_rear = %d, Q_size = %d\n", Single_values->Q_front, Single_values->Q_next_front, Single_values->Q_rear, Q_size);
                }

                printf("[HI][5]\n");
                printf("compSize = %d\n", compSize);
                contributeDistToEachNodes<<<(compSize + 1023)/1024, 1024>>>(g_dist_s, g_CCs, g_mapNodeID_New_to_Old, g_newID_infos_ff, g_newID_infos_r, g_comp_newCsrOffset, g_newNodesCompID, g_newNodesID_arr, compSize, sourceNewID, g_Single_values);
                hipDeviceSynchronize();
                printf("[HI][6]\n");
            }
            else{
                mappingCount = 0;
                register int new_nID = -1;
                register int old_nID = -1;
                for(int new_nidx = _csr->orderedCsrV[sourceNewID] ; new_nidx < _csr->orderedCsrV[sourceNewID + 1] ; new_nidx ++){
                    new_nID = _csr->orderedCsrE[new_nidx];
                    old_nID = _csr->mapNodeID_New_to_Old[new_nID];
                    if(nodeDone[new_nID] == 0 && (old_nID <= oriEndNodeID)){
                        nodeDone[new_nID] = 1;
                        SI[new_nID] = 1 << mappingCount;
                        mapNodes[mappingCount] = new_nID;
                        
                        printf("\tshared new_nID %d, old_nID %d, SI = %x\n", new_nID, old_nID, SI[new_nID]);

                        mappingCount ++;
                        if(mappingCount == 32){
                            break;
                        }
                    }
                }
                
                hipMemcpy(g_mapNodes, mapNodes, sizeof(int) * 32, hipMemcpyHostToDevice); //Host 的 mapNodes 只有前mappingCount個有 nodes, 之後的 cell 都沒有nodes，剛好reset g_mapNodes
                copySI_toDevice<<<1, 32>>>(g_mapNodes, g_SI, mappingCount);
                // hipMemcpyAsync(g_SI, SI, sizeof(unsigned int) * (_csr->csrVSize * 2), hipMemcpyHostToDevice); //因為Host的SI只有那些mapNodes有值，其他SI都是0，剛好reset了g_SI。
                // hipDeviceSynchronize();

                #pragma region SourceTraverse
                while(Q_size > 0){
                    hipMemcpy(g_Single_values, Single_values, sizeof(struct Single_values), hipMemcpyDeviceToHost);

                }
                #pragma endregion SourceTraverse
            }
            
            exit(1);
        }   
    }
    #pragma endregion Traverse


}

int main(int argc, char* argv[]){
    char* datasetPath = argv[1];
    printf("exeName = %s\n", argv[0]);
    printf("datasetPath = %s\n", datasetPath);
    struct Graph* graph = buildGraph(datasetPath);
    struct CSR* csr     = createCSR(graph);
    int* CCs = (int*)calloc(sizeof(int), csr->csrVSize);

    preprocess_then_parallel_sharedBased_DegreeOrder(csr, CCs);


}

void quicksort_nodeID_with_degree(int* _nodes, int* _nodeDegrees, int _left, int _right){
    if(_left > _right){
        return;
    }
    int smallerAgent = _left;
    int smallerAgentNode = -1;
    int equalAgent = _left;
    int equalAgentNode = -1;
    int largerAgent = _right;
    int largerAgentNode = -1;

    int pivotNode = _nodes[_right];
    // printf("pivot : degree[%d] = %d .... \n", pivotNode, _nodeDegrees[pivotNode]);
    int tempNode = 0;
    while(equalAgent <= largerAgent){
        #ifdef DEBUG
        // printf("\tsmallerAgent = %d, equalAgent = %d, largerAgent = %d\n", smallerAgent, equalAgent, largerAgent);
        #endif

        smallerAgentNode = _nodes[smallerAgent];
        equalAgentNode = _nodes[equalAgent];
        largerAgentNode = _nodes[largerAgent];
        
        #ifdef DEBUG
        // printf("\tDegree_s[%d] = %d, Degree_e[%d] = %d, Degree_l[%d] = %d\n", smallerAgentNode, _nodeDegrees[smallerAgentNode], equalAgentNode, _nodeDegrees[equalAgentNode], largerAgentNode, _nodeDegrees[largerAgentNode]);
        #endif

        if(_nodeDegrees[equalAgentNode] < _nodeDegrees[pivotNode]){ //equalAgentNode的degree < pivotNode的degree
            // swap smallerAgentNode and equalAgentNode
            tempNode = _nodes[smallerAgent];
            _nodes[smallerAgent] = _nodes[equalAgent];
            _nodes[equalAgent] = tempNode;

            smallerAgent ++;
            equalAgent ++;
        }
        else if(_nodeDegrees[equalAgentNode] > _nodeDegrees[pivotNode]){ //equalAgentNode的degree > pivotNode的degree
            // swap largerAgentNode and equalAgentNode
            tempNode = _nodes[largerAgent];
            _nodes[largerAgent] = _nodes[equalAgent];
            _nodes[equalAgent] = tempNode;

            largerAgent --;
        }
        else{ //equalAgentNode的degree == pivotNode的degree
            equalAgent ++;
        }

    }
    
    // exit(1);
    #ifdef DEBUG
        
    #endif

    // smallerAgent現在是pivot key的開頭
    // largerAgent現在是pivotKey的結尾
    quicksort_nodeID_with_degree(_nodes, _nodeDegrees, _left, smallerAgent - 1);
    quicksort_nodeID_with_degree(_nodes, _nodeDegrees, largerAgent + 1, _right);
}

void sortEachComp_NewID_with_degree(struct CSR* _csr, int* _newNodesID_arr, int* _newNodesDegree_arr){
    /**
     * 1. assign newID to _newNodesID_arr
     * 2. assign degree according to oldID of newID to _newNodesDegree_arr
    */
    for(int newID = 0 ; newID <= _csr->newEndID ; newID ++){
        _newNodesID_arr[newID]       = newID;
        _newNodesDegree_arr[newID]   = _csr->orderedCsrV[newID + 1] - _csr->orderedCsrV[newID];
        
        // printf("newID %d, oldID %d, degree %d\n", _newNodesID_arr[newID], _csr->mapNodeID_New_to_Old[newID], _newNodesDegree_arr[newID]);
    }

    /**
     * 在每個 component內 依照degree進行排序
    */
    for(int compID = 0 ; compID <= _csr->compEndID ; compID ++){
        int compSize = _csr->comp_newCsrOffset[compID + 1] - _csr->comp_newCsrOffset[compID];
        // printf("compID %d, compSize %d\n", compID, compSize);
        quicksort_nodeID_with_degree(_newNodesID_arr, _newNodesDegree_arr, _csr->comp_newCsrOffset[compID], _csr->comp_newCsrOffset[compID + 1] - 1);
    }

    // for(int newID_idx = 0 ; newID_idx <= _csr->newEndID ; newID_idx ++){
    //     int newID = _newNodesID_arr[newID_idx];
    //     int degree = _csr->orderedCsrV[newID + 1] - _csr->orderedCsrV[newID];
    //     printf("newID %d, degree %d, compID %d\n", newID, degree, _csr->newNodesCompID[newID]);
    // }
}


__global__ void g_newID_infos_AOS_to_SOA(struct newID_info* _g_newID_infos_AOS, int* _g_newID_infos_ff, int* _g_newID_infos_r, struct Single_values* _g_csr_values){
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < (_g_csr_values->newEndID + 1)){
        
        _g_newID_infos_ff[tid]  = __ldg(&(_g_newID_infos_AOS[tid].ff));
        _g_newID_infos_r[tid]   = __ldg(&(_g_newID_infos_AOS[tid].w));
    }
}

__global__ void resetData(int* _g_dist_s, int* _g_Q, unsigned int* _g_SI, unsigned int* _g_R, struct Single_values* _g_single_values, int sourceNewID){
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < (_g_single_values->csrVSize * 2)){
        _g_dist_s[tid]  = -1;
        _g_SI[tid]      = 0;
        _g_R[tid]       = 0;
    }
    if(tid == 0){
        _g_dist_s[sourceNewID]              = 0;
        _g_single_values->Q_front           = 0;
        _g_single_values->Q_next_front      = 1;
        _g_single_values->Q_rear            = 1; 
        printf("g_Q_front = %d, g_Q_rear = %d, ", _g_single_values->Q_next_front, _g_single_values->Q_rear);
        _g_Q[0] = sourceNewID; //enqueue
        printf("_g_Q[0] = %d, _g_dist_s[%d] = %d\n", _g_Q[0], sourceNewID, _g_dist_s[sourceNewID]);
        _g_single_values->mappingCount  = 0;
    }
}

__global__ void mappingCount(int* _g_orderedCsrV, int* _g_orderedCsrE, int* _g_mapNodeID_New_to_Old, int* _g_nodeDone, struct Single_values* _g_single_values, int sourceNewID){
    const int tid           = threadIdx.x + blockIdx.x * blockDim.x;
    register int new_nidx   = tid + __ldg(&(_g_orderedCsrV[sourceNewID]));
    if(new_nidx < _g_orderedCsrV[sourceNewID + 1]){
        register int new_nID = _g_orderedCsrE[new_nidx];
        register int old_nID = _g_mapNodeID_New_to_Old[new_nID];
        if((_g_nodeDone[new_nID] == 0) && (old_nID <= _g_single_values->oriEndNodeID)){
            int old_mappingCount = atomicAdd(&(_g_single_values->mappingCount), 1);
        }
    }
    if(tid == 0){
        _g_nodeDone[sourceNewID] = 1;
    }
}

__global__ void ordinaryTraverseOneLevel(int* _g_orderedCsrV, int* _g_orderedCsrE, int* _g_mapNodeID_New_to_Old, int* _g_dist_s, int* _g_Q, struct Single_values* _g_single_values, int _Q_size){
    register const int blockID = blockIdx.x; 
    // printf("bID = %d\n", blockID);
    if(blockID >= _Q_size){
        return;
    }
    
    register const int block_CurNode_Idx    = blockID + _g_single_values->Q_front;
    register const int block_CurNode_NewID  = __ldg(&(_g_Q[block_CurNode_Idx])); //整個block都在訪問CurNode_NewID的neighbors
    register const int degree               = _g_orderedCsrV[block_CurNode_NewID + 1] - _g_orderedCsrV[block_CurNode_NewID];
    //如果一個node有180個neighbor, blockDim.x = 96的話, neighborOffset = 2 代表這個block要用96個thread做兩輪,才可以把neighbor都訪問完
    register const int neighborOffset = (int)ceil(degree/(blockDim.x * 1.0)); 
    

    for(int i = 0 ; i < neighborOffset ; i ++){
        register const int thread_neighbor_idx      = _g_orderedCsrV[block_CurNode_NewID] + threadIdx.x + i * blockDim.x;
        register const int thread_neighbor_NewID    = __ldg(&(_g_orderedCsrE[thread_neighbor_idx]));
        // printf("blockID %d, threadID %d, block_CurNodeIdx = %d, block_CurNode_NewID = %d, thread_neighbor_idx = %d, thread_neighbor_NewID = %d\n", blockID, threadIdx.x, block_CurNode_Idx, block_CurNode_NewID, thread_neighbor_idx, thread_neighbor_NewID);
        if(thread_neighbor_idx < _g_orderedCsrV[block_CurNode_NewID + 1] && (atomicCAS(&(_g_dist_s[thread_neighbor_NewID]), -1, _g_dist_s[block_CurNode_NewID] + 1) == -1)){
            
            int enQ_position    = atomicAdd(&(_g_single_values->Q_rear), 1);
            _g_Q[enQ_position]  = thread_neighbor_NewID;
            printf("[EnQ] block_CurNode_NewID = %d, Q_pos = %d, \tthread_neighbor_newID = %d, \tdist = %d\n", block_CurNode_NewID, enQ_position, thread_neighbor_NewID, atomicAdd(&(_g_dist_s[thread_neighbor_NewID]), 0));
        }
    }
}

__global__ void contributeDistToEachNodes(int* _g_dist_s, int* _g_CCs, int* _g_mapNodeID_New_to_Old, int* _g_newID_infos_ff, int* _g_newID_infos_r, int* _g_comp_newCsrOffset, int* _g_newNodesCompID, int* _g_newNodesID_arr, int _comp_size, int _sourceNewID, struct Single_values* _g_Single_values){
    register const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // int laneID = threadIdx.x % 32;

    __shared__ int sharedData[2]; //sharedData[0] 是 sourceOldID, sharedData[1] 是 sourceOldID 的 compID
    if(tid == 0){
        sharedData[0] = __ldg(&(_g_mapNodeID_New_to_Old[_sourceNewID]));
        sharedData[1] = __ldg(&(_g_newNodesCompID[_sourceNewID]));;
    }
    __syncthreads();

    // if(laneID == 0){ //為了不要用太多的 __ldg
    //     sourceOldID = __ldg(&(_g_mapNodeID_New_to_Old[_sourceNewID])); // sourceNewID 的 OldID，用於在_g_CCs賦值
    //     compID      = __ldg(&(_g_newNodesCompID[_sourceNewID])); // sourceNewID 在的 compID
    // }
    // register int sharedSourceOldID =  __shfl_sync(0xffffffff, sourceOldID, 0);
    // register int sharedCompID = __shfl_sync(0xffffffff, compID, 0);

    // register const int SourceOldID = __ldg(&(_g_mapNodeID_New_to_Old[_sourceNewID])); // sourceNewID 的 OldID，用於在_g_CCs賦值
    // int compID = __ldg(&(_g_newNodesCompID[_sourceNewID]));  // sourceNewID 在的 compID
    // __syncthreads();

    if(tid < _comp_size){
        int compOtherNodeNew_idx = tid + __ldg(&(_g_comp_newCsrOffset[sharedData[1]]));
        register const int compOtherNodeNewID = __ldg(&(_g_newNodesID_arr[compOtherNodeNew_idx]));
        register const int compOtherNodeOldID = __ldg(&(_g_mapNodeID_New_to_Old[compOtherNodeNewID]));
        
        if(compOtherNodeOldID > _g_Single_values->oriEndNodeID){ //代表這個compOtherNodeOldID 是 apclone
            return;
        }
        _g_CCs[compOtherNodeOldID] += _g_newID_infos_r[_sourceNewID] * _g_dist_s[compOtherNodeNewID] + _g_newID_infos_ff[_sourceNewID];
        printf("tid = %d,\t_g_CCs[old = %d, new = %d] = %d,\t g_dist_s[new = %d] = %d\n", tid, compOtherNodeOldID, compOtherNodeNewID, _g_CCs[compOtherNodeOldID], compOtherNodeNewID, _g_dist_s[compOtherNodeNewID]);
    }
}

__global__ void copySI_toDevice(int* _g_mapNodes, unsigned int* _g_SI, int _mappingCount){
    register const int tid = threadIdx.x; //這個kernel只用一個block，該block只有32個threads.
    if(tid < _mappingCount){
        
        register const int thread_mapped_newID = __ldg(&(_g_mapNodes[tid]));
        register const int SI_value = 1 << tid;
        
        _g_SI[thread_mapped_newID] = SI_value;
        printf("tid = %d, _g_mapNodes[%d] = %d, SI_value = %x, g_SI[%d] = %x\n", tid, tid, _g_mapNodes[tid], SI_value, thread_mapped_newID, _g_SI[thread_mapped_newID]);
    }
}

__global__ void First_sharedTraverseOneLevel(int* _g_orderedCsrV, int* _g_orderedCsrE, int* _g_dist_s, int* _g_Q, struct Single_values* _g_single_values, int _Q_size, int* _g_SI, int* _g_R){
    register const int blockID = blockIdx.x;

    if(blockID >= _Q_size){
        return;
    }

    register const int block_CurNode_Idx    = blockID + _g_single_values->Q_front;
    register const int block_CurNode_NewID  = __ldg(&(_g_Q[block_CurNode_Idx]));
    register const int degree               = _g_orderedCsrV[block_CurNode_NewID + 1] - _g_orderedCsrV[block_CurNode_NewID];

    register const int neighborOffset = (int)ceil(degree/(blockDim.x * 1.0));

    for(int i = 0 ; i < neighborOffset ; i ++){
        register const int thread_neighbor_idx      = _g_orderedCsrV[block_CurNode_NewID] + threadIdx.x + i * blockDim.x;
        register const int thread_neighbor_NewID    = __ldg(&(_g_orderedCsrE[thread_neighbor_idx]));
        if(thread_neighbor_idx < _g_orderedCsrV[block_CurNode_NewID + 1]){
            
        }
    }
}